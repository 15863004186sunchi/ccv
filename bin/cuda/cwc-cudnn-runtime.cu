#include "hip/hip_runtime.h"
#undef USE_DISPATCH // nvcc doesn't support libdispatch
extern "C" {
#include "ccv.h"
}
#include <ctype.h>
#include <hipDNN.h>
#include "../lib/ccv_convnet.c"

typedef struct {
	hipdnnTensorDescriptor_t tensor;
	hipdnnFilterDescriptor_t filter;
	hipdnnConvolutionDescriptor_t convolution;
	hipdnnConvolutionFwdAlgo_t forwards;
} cwc_cudnn_layer_t;

extern "C" void cwc_cudnn_runtime(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_convnet_train_param_t params)
{
	int batch = params.mini_batch;
	int i, j;
	const int device_id = 0;
	_cwc_convnet_alloc_reserved_both(convnet, batch, 1, params.layer_params);
	cwc_convnet_context_t* context = GPU(convnet)->contexts;
	for (i = 0; i < convnet->rows * convnet->cols * convnet->channels; i++)
		convnet->mean_activity->data.f32[i] = 128;
	cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, 0, ccv_size(225, 225), 225, 225, convnet->rows, convnet->cols, convnet->channels, 1000, 0, batch, 0, batch, context->host[device_id].input, context->host[device_id].c);
	hipMemcpy(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * batch, hipMemcpyHostToDevice);

	ccv_convnet_t* update_params = _ccv_convnet_update_new(convnet);
	_ccv_convnet_update_zero(update_params);

	// first convolutional layer forward propagate
	ccv_convnet_layer_t* first_gpu_layer = GPU(convnet)->device[device_id].layers;
	// these are the setups for TITAN, thus, skip the benching phase
	EXTRA(first_gpu_layer)->vary.convolutional.forward.x = 4;
	EXTRA(first_gpu_layer)->vary.convolutional.forward.y = 8;
	EXTRA(first_gpu_layer)->vary.convolutional.forward.z = 32;
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, context->device[device_id].data_stream);
	cwc_convnet_convolutional_forward_propagate(first_gpu_layer, first_gpu_layer->input.matrix.rows, first_gpu_layer->input.matrix.cols, batch, context->device[device_id].input, GPU(convnet)->device[device_id].forwards[0], context->device[device_id].data_stream);
	hipEventRecord(stop, context->device[device_id].data_stream);
	hipEventSynchronize(stop);
	float elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipStreamSynchronize(context->device[device_id].data_stream);
	printf("%d %d %d, elapsed time for first convolutional layer fprop: %f milliseconds\n", EXTRA(first_gpu_layer)->vary.convolutional.forward.x, EXTRA(first_gpu_layer)->vary.convolutional.forward.y, EXTRA(first_gpu_layer)->vary.convolutional.forward.z, elapsed_time);
	int first_out_rows, first_out_cols, first_out_partition, first_out_channels = first_gpu_layer->net.convolutional.count;
	ccv_convnet_make_output(first_gpu_layer, first_gpu_layer->input.matrix.rows, first_gpu_layer->input.matrix.cols, &first_out_rows, &first_out_cols, &first_out_partition);
	float* first_out = 0;
	hipHostMalloc(&first_out, sizeof(float) * first_out_rows * first_out_cols * first_out_channels * batch);
	hipMemcpy(first_out, GPU(convnet)->device[device_id].forwards[0], sizeof(float) * first_out_rows * first_out_cols * first_out_channels * batch, hipMemcpyDeviceToHost);
	printf("finished forward propagate first convolutional layer on GPU\n");

	int x, y, k, c;
	for (i = 0; i < batch; i++)
	{
		printf("doing batch %d of %d\n", i + 1, batch);
		ccv_categorized_t* categorized = (ccv_categorized_t*)ccv_array_get(categorizeds, i);
		for (x = 0; x < categorized->matrix->rows * categorized->matrix->cols * CCV_GET_CHANNEL(categorized->matrix->type); x++)
			categorized->matrix->data.f32[x] = categorized->matrix->data.f32[x] - 128;

		// first convolutional layer forward propagate
		ccv_convnet_layer_t* first_cpu_layer = convnet->layers;
		_ccv_convnet_convolutional_forward_propagate(first_cpu_layer, categorized->matrix, convnet->acts);
		ccv_dense_matrix_t* a = convnet->acts[0];
		for (y = 0; y < first_out_rows; y++)
			for (x = 0; x < first_out_cols; x++)
				for (k = 0; k < first_out_channels; k++)
				{
					float p = first_out[k * first_out_rows * first_out_cols * batch + (y * first_out_cols + x) * batch + i];
					float q = a->data.f32[y * first_out_cols * first_out_channels + x * first_out_channels + k];
					float delta = fabs(p - q) / ccv_max(ccv_max(fabs(p), fabs(q)), 1);
					if (delta > 1e-4)
						printf("conv fprop 1: %d %d %d %d: |%f - %f| = %f\n", i, x, y, k, p, q, delta);
				}
	}
}
