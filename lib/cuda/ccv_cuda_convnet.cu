#include "hip/hip_runtime.h"
extern "C" {
#include "ccv_cuda_convnet.h"
}

static inline void _ccv_convnet_compute_output_scale(int a_rows, int a_cols, ccv_convnet_layer_t* layer, int* rows, int* cols)
{
	assert(rows != 0 && cols != 0);
	switch(layer->type)
	{
		case CCV_CONVNET_CONVOLUTIONAL:
			assert(layer->net.convolutional.rows % 2); // as of now, don't support even number of kernel size
			assert(layer->net.convolutional.cols % 2);
			assert((a_rows + layer->net.convolutional.border * 2 - layer->net.convolutional.rows) % layer->net.convolutional.strides == 0);
			assert((a_cols + layer->net.convolutional.border * 2 - layer->net.convolutional.cols) % layer->net.convolutional.strides == 0);
			*rows = (a_rows + layer->net.convolutional.border * 2 - layer->net.convolutional.rows) / layer->net.convolutional.strides + 1;
			*cols = (a_cols + layer->net.convolutional.border * 2 - layer->net.convolutional.cols) / layer->net.convolutional.strides + 1;
			break;
		case CCV_CONVNET_FULL_CONNECT:
			*rows = layer->net.full_connect.count;
			*cols = 1;
			break;
		case CCV_CONVNET_MAX_POOL:
		case CCV_CONVNET_AVERAGE_POOL:
			assert((a_rows - layer->net.pool.size) % layer->net.pool.strides == 0);
			assert((a_cols - layer->net.pool.size) % layer->net.pool.strides == 0);
			*rows = (a_rows - layer->net.pool.size) / layer->net.pool.strides + 1;
			*cols = (a_cols - layer->net.pool.size) / layer->net.pool.strides + 1;
			break;
	}
}

template <int input_per_thread, int filter_per_thread, int first_pass>
__global__ void _ccv_kern_convolutional_forward_propagate(const int strides, const int border,
		float* input, const int rows, const int cols, const int channels,
		float* out, const int out_rows, const int out_cols,
		float* filter, const int filter_rows, const int filter_cols, const int count)
{
	// gridDim.x == out_rows
	// gridDim.y == out_cols
	extern __shared__ float shared[];
	float* shared_block = &shared[0];
	float* shared_filter = &shared[blockDim.x * input_per_thread];
	float prod[filter_per_thread][input_per_thread];
	const int thidx = threadIdx.x + threadIdx.y * blockDim.x;
	const int thcnt = blockDim.x * blockDim.y;
	const int input_loads = input_per_thread / blockDim.y;
	const int filter_loads = filter_per_thread / blockDim.x;
	const int batch = input_per_thread * blockDim.x;
	int i, j, x, y;
	for (i = 0; i < filter_per_thread; i++)
		for (j = 0; j < input_per_thread; j++)
			prod[i][j] = 0;
	input += (blockIdx.x * strides * cols + blockIdx.y * strides) * batch;
	for (y = 0; y < filter_rows; y++)
		for (x = 0; x < filter_cols; x++)
		{
			for (i = 0; i < input_loads; i++)
				shared_block[i * thcnt + thidx] = input[(y * cols + x) * batch + i * thcnt + thidx];
			for (i = 0; i < filter_loads; i++)
				shared_filter[i * thcnt + thidx] = filter[(y * filter_cols + x) * count + i * thcnt + thidx];
			__syncthreads();
			for (i = 0; i < filter_per_thread; i++)
				for (j = 0; j < input_per_thread; j++)
					prod[i][j] += shared_block[j + threadIdx.x * input_per_thread] * shared_filter[i + threadIdx.y * filter_per_thread];
			__syncthreads();
		}
	const int outcnt = out_rows * out_cols * batch;
	out += (blockIdx.x * out_cols + blockIdx.y) * batch;
	if (first_pass)
		for (i = 0; i < filter_per_thread; i++)
			for (j = 0; j < input_per_thread; j++)
				out[(i + threadIdx.y * filter_per_thread) * outcnt + j + threadIdx.x * input_per_thread] = prod[i][j];
	else
		for (i = 0; i < filter_per_thread; i++)
			for (j = 0; j < input_per_thread; j++)
				out[(i + threadIdx.y * filter_per_thread) * outcnt + j + threadIdx.x * input_per_thread] += prod[i][j];
}

#include <sys/time.h>
#include <ctype.h>

static unsigned int get_current_time(void)
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000 + tv.tv_usec / 1000;
}

static void _ccv_cu_convolutional_forward_propagate(ccv_convnet_layer_t* layer, int batch, int rows, int cols, int ch, float* a, float* d, float** b)
{
	int out_rows, out_cols;
	_ccv_convnet_compute_output_scale(rows, cols, layer, &out_rows, &out_cols);
	assert(b);
	float* db = *b;
	if (db)
		hipMalloc(&db, sizeof(float) * out_rows * out_cols * layer->net.convolutional.count * batch);
	*b = db;
	float* od_w = 0;
	hipMalloc(&od_w, sizeof(float) * layer->wnum);
	hipMemcpy(od_w, layer->w, sizeof(float) * layer->wnum, hipMemcpyHostToDevice);
	dim3 threads_per_block(batch / 32, layer->net.convolutional.count / 32);
	dim3 num_blocks(out_rows, out_cols);
	unsigned int elapsed_time = get_current_time();
	_ccv_kern_convolutional_forward_propagate
		<32, 32, 1>
		<<<num_blocks, threads_per_block, sizeof(float) * (batch + layer->net.convolutional.count)>>>
		(layer->net.convolutional.strides, layer->net.convolutional.border,
		 a, rows, cols, ch,
		 db, out_rows, out_cols,
		 od_w, layer->net.convolutional.rows, layer->net.convolutional.cols, layer->net.convolutional.count);
	int i;
	for (i = 1; i < ch; i++)
		_ccv_kern_convolutional_forward_propagate
			<32, 32, 0>
			<<<num_blocks, threads_per_block, sizeof(float) * (batch + layer->net.convolutional.count)>>>
			(layer->net.convolutional.strides, layer->net.convolutional.border,
			 a + i * batch * rows * cols, rows, cols, ch,
			 db + i * batch * out_rows * out_cols, out_rows, out_cols,
			 od_w + i * layer->net.convolutional.rows * layer->net.convolutional.cols, layer->net.convolutional.rows, layer->net.convolutional.cols, layer->net.convolutional.count);
	elapsed_time = get_current_time() - elapsed_time;
	printf("cuda elapsed time: %u\n", elapsed_time);
}

static void _ccv_convnet_convolutional_forward_propagate(ccv_convnet_layer_t* layer, ccv_dense_matrix_t* a, ccv_dense_matrix_t* d, ccv_dense_matrix_t** b)
{
	int rows, cols;
	_ccv_convnet_compute_output_scale(a->rows, a->cols, layer, &rows, &cols);
	int ch = layer->net.convolutional.channels;
	int count = layer->net.convolutional.count;
	int strides = layer->net.convolutional.strides;
	int border = layer->net.convolutional.border;
	int kernel_rows = layer->net.convolutional.rows;
	int kernel_cols = layer->net.convolutional.cols;
	int type = CCV_32F | count;
	assert(CCV_GET_CHANNEL(a->type) == ch);
	assert(CCV_GET_DATA_TYPE(a->type) == CCV_32F);
	ccv_dense_matrix_t* db = *b = ccv_dense_matrix_renew(*b, rows, cols, type, type, 0);
	int i, j, x, y, k;
#define for_block(act_block_setup, act_block_begin, act_block_end) \
	for (k = 0; k < count; k++) \
	{ \
		float* ap = a->data.f32; \
		float* bp = db->data.f32 + k; \
		float* layer_w = layer->w + k * kernel_rows * kernel_cols * ch; \
		float bias = layer->bias[k]; \
		act_block_setup; \
		for (i = 0; i < db->rows; i++) \
		{ \
			int comy = ccv_max(i * strides - border, 0) - (i * strides - border); \
			int maxy = kernel_rows - comy - (i * strides + kernel_rows - ccv_min(a->rows + border, i * strides + kernel_rows)); \
			comy *= ch * kernel_cols; \
			for (j = 0; j < db->cols; j++) \
			{ \
				act_block_begin; \
				float v = bias; \
				int comx = (ccv_max(j * strides - border, 0) - (j * strides - border)) * ch; \
				int maxx = kernel_cols * ch - comx - (j * strides + kernel_cols - ccv_min(a->cols + border, j * strides + kernel_cols)) * ch; \
				float* w = layer_w + comx + comy; \
				float* apz = ap + ccv_max(j * strides - border, 0) * ch; \
				/* when we have border, we simply do zero padding */ \
				for (y = 0; y < maxy; y++) \
				{ \
					for (x = 0; x < maxx; x++) \
						v += w[x] * apz[x]; \
					w += kernel_cols * ch; \
					apz += a->cols * ch; \
				} \
				bp[j * count] = ccv_max(0, v) /* ReLU */; \
				act_block_end; \
			} \
			bp += db->cols * count; \
			ap += a->cols * ch * (ccv_max((i + 1) * strides - border, 0) - ccv_max(i * strides - border, 0)); \
		} \
	}
	if (d)
	{
#define act_block_setup \
		int* dp = d->data.i32 + k;
#define act_block_begin \
		if (!*dp) \
		{
#define act_block_end \
		} else \
			bp[j * count] = 0; \
		dp += count;
		for_block(act_block_setup, act_block_begin, act_block_end);
#undef act_block_setup
#undef act_block_begin
#undef act_block_end
	} else {
		for_block(/* empty act block setup */, /* empty act block begin */, /* empty act block end */);
	}
#undef for_block
}

void ccv_cu_convnet_encode(ccv_convnet_t* convnet, ccv_dense_matrix_t** a, ccv_dense_matrix_t** b, int batch)
{
	int ch = CCV_GET_CHANNEL(a[0]->type);
	int rows = a[0]->rows, cols = a[0]->cols;
	float* vec = 0;
	hipHostMalloc(&vec, sizeof(float) * batch * rows * cols * ch);
	int i, j, k;
	for (i = 0; i < batch; i++)
		for (k = 0; k < ch; k++)
			for (j = 0; j < rows * cols; j++)
				vec[i + (k * rows * cols + j) * batch] = a[i]->data.f32[j * ch + k];
	float* od_vec = 0;
	hipMalloc(&od_vec, sizeof(float) * batch * rows * cols * ch);
	hipMemcpy(od_vec, vec, sizeof(float) * batch * rows * cols * ch, hipMemcpyHostToDevice);
	float* od_out = 0;
	_ccv_cu_convolutional_forward_propagate(convnet->layers, batch, rows, cols, ch, od_vec, 0, &od_out);
	int out_rows, out_cols;
	_ccv_convnet_compute_output_scale(rows, cols, convnet->layers, &out_rows, &out_cols);
	float* out = 0;
	hipHostMalloc(&out, sizeof(float) * batch * out_rows * out_cols * convnet->layers->net.convolutional.count);
	hipMemcpy(out, od_out, sizeof(float) * batch * out_rows * out_cols * convnet->layers->net.convolutional.count, hipMemcpyDeviceToHost);
	unsigned int elapsed_time = get_current_time();
	for (i = 0; i < batch; i++)
	{
		ccv_dense_matrix_t* b = 0;
		_ccv_convnet_convolutional_forward_propagate(convnet->layers, a[i], 0, &b);
		int x, y, ch = convnet->layers->net.convolutional.count;
		for (k = 0; k < ch; k++)
			for (y = 0; y < b->rows; y++)
				for (x = 0; x < b->cols; x++)
				{
					float delta = fabsf(out[k * out_rows * out_cols * batch + (x + y * out_cols) * batch + i] - b->data.f32[(x + y * out_cols) * ch + k]);
					if (delta > 1e-3)
						printf("%d %g %g\n", i, out[k * out_rows * out_cols * batch + (x + y * out_cols) * batch + i], b->data.f32[(x + y * out_cols) * ch + k]);
				}
		ccv_matrix_free(b);
	}
	elapsed_time = get_current_time() - elapsed_time;
	printf("cpu elapsed time: %u\n", elapsed_time);
}

void ccv_cu_convnet_classify(ccv_convnet_t* convnet, ccv_dense_matrix_t** a, int* labels, int batch)
{
}

void ccv_cu_convnet_supervised_train(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_array_t* tests, ccv_convnet_train_param_t params)
{
	assert(categorizeds->rnum >= 32);
	int i;
	ccv_dense_matrix_t* a[32];
	for (i = 0; i < 32; i++)
	{
		ccv_categorized_t* categorized = (ccv_categorized_t*)ccv_array_get(categorizeds, i);
		ccv_dense_matrix_t* image = 0;
		ccv_read(categorized->file.filename, &image, CCV_IO_ANY_FILE | CCV_IO_RGB_COLOR);
		ccv_dense_matrix_t* b = 0;
		if (image->rows > 251 && image->cols > 251)
			ccv_resample(image, &b, 0, ccv_max(251, (int)(image->rows * 251.0 / image->cols + 0.5)), ccv_max(251, (int)(image->cols * 251.0 / image->rows + 0.5)), CCV_INTER_AREA);
		else if (image->rows < 251 || image->cols < 251)
			ccv_resample(image, &b, 0, ccv_max(251, (int)(image->rows * 251.0 / image->cols + 0.5)), ccv_max(251, (int)(image->cols * 251.0 / image->rows + 0.5)), CCV_INTER_CUBIC);
		else
			b = image;
		if (b != image)
			ccv_matrix_free(image);
		ccv_dense_matrix_t* c = 0;
		ccv_slice(b, (ccv_matrix_t**)&c, CCV_32F, 0, 0, 225, 225);
		int j, ch = CCV_GET_CHANNEL(c->type);
		for (j = 0; j < c->rows * c->cols * ch; j++)
			c->data.f32[j] = c->data.f32[j] / 255.0 * 2 - 1;
		a[i] = c;
		ccv_matrix_free(b);
	}
	ccv_cu_convnet_encode(convnet, a, 0, 32);
}
