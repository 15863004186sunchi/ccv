#include "hip/hip_runtime.h"
extern "C" {
#include "cog.h"
}

// this structure holds intermediate on-device memory representation of convnet
typedef struct {
	ccv_convnet_layer_t* layers;
	ccv_convnet_layer_t* updates;
} cog_convnet_t;

#define GPU(x) ((cog_convnet_t*)((x)->reserved))

static inline void _ccv_convnet_compute_output_scale(int a_rows, int a_cols, ccv_convnet_layer_t* layer, int* rows, int* cols)
{
	assert(rows != 0 && cols != 0);
	switch(layer->type)
	{
		case CCV_CONVNET_CONVOLUTIONAL:
			assert(layer->net.convolutional.rows % 2); // as of now, don't support even number of kernel size
			assert(layer->net.convolutional.cols % 2);
			assert((a_rows + layer->net.convolutional.border * 2 - layer->net.convolutional.rows) % layer->net.convolutional.strides == 0);
			assert((a_cols + layer->net.convolutional.border * 2 - layer->net.convolutional.cols) % layer->net.convolutional.strides == 0);
			*rows = (a_rows + layer->net.convolutional.border * 2 - layer->net.convolutional.rows) / layer->net.convolutional.strides + 1;
			*cols = (a_cols + layer->net.convolutional.border * 2 - layer->net.convolutional.cols) / layer->net.convolutional.strides + 1;
			break;
		case CCV_CONVNET_FULL_CONNECT:
			*rows = layer->net.full_connect.count;
			*cols = 1;
			break;
		case CCV_CONVNET_MAX_POOL:
		case CCV_CONVNET_AVERAGE_POOL:
			assert((a_rows - layer->net.pool.size) % layer->net.pool.strides == 0);
			assert((a_cols - layer->net.pool.size) % layer->net.pool.strides == 0);
			*rows = (a_rows - layer->net.pool.size) / layer->net.pool.strides + 1;
			*cols = (a_cols - layer->net.pool.size) / layer->net.pool.strides + 1;
			break;
	}
}

template <int input_per_thread, int filter_per_thread>
__global__ void _cog_kern_convolutional_forward_propagate(const int strides, const int border, const int batch,
		float* input, const int rows, const int cols, const int channels,
		float* out, const int out_rows, const int out_cols,
		float* filter, const int filter_rows, const int filter_cols, const int count,
		float* const biases)
{
	// gridDim.x == out_rows
	// gridDim.y == out_cols
	extern __shared__ float shared[];
	float* shared_block = &shared[0];
	assert(batch == input_per_thread * blockDim.x);
	float* shared_weights = &shared[batch];
	assert(count == filter_per_thread * blockDim.y);
	float* shared_bias = &shared[batch + count];
	float prod[filter_per_thread][input_per_thread];
	const int thidx = threadIdx.x + threadIdx.y * blockDim.x;
	const int thcnt = blockDim.x * blockDim.y;
	const int input_loads = (batch + thcnt - 1) / thcnt;
	const int filter_loads = (count + thcnt - 1) / thcnt;
	int c, i, j, x, y;
	#pragma unroll
	for (i = 0; i < filter_per_thread; i++)
		#pragma unroll
		for (j = 0; j < input_per_thread; j++)
			prod[i][j] = 0;
	input += (blockIdx.x * strides * cols + blockIdx.y * strides) * batch;
	#pragma unroll
	for (i = 0; i < filter_loads; i++)
		if (i * thcnt + thidx < count)
			shared_bias[i * thcnt + thidx] = biases[i * thcnt + thidx];
	for (c = 0; c < channels; c++)
	{
		for (y = 0; y < filter_rows; y++)
		{
			const int iy = y + blockIdx.x * strides - border;
			for (x = 0; x < filter_cols; x++)
			{
				const int ix = x + blockIdx.y * strides - border;
				if (iy >= 0 && iy < rows && ix >= 0 && ix < cols)
				{
					#pragma unroll
					for (i = 0; i < input_loads; i++)
						if (i * thcnt + thidx < batch)
							shared_block[i * thcnt + thidx] = input[((y - border) * cols + x - border) * batch + i * thcnt + thidx];
					#pragma unroll
					for (i = 0; i < filter_loads; i++)
						if (i * thcnt + thidx < count)
							shared_weights[i * thcnt + thidx] = filter[(y * filter_cols + x) * count + i * thcnt + thidx];
					__syncthreads();
					#pragma unroll
					for (i = 0; i < filter_per_thread; i++)
						#pragma unroll
						for (j = 0; j < input_per_thread; j++)
							prod[i][j] += shared_block[j + threadIdx.x * input_per_thread] * shared_weights[i + threadIdx.y * filter_per_thread];
					__syncthreads();
				}
			}
		}
		input += rows * cols * batch;
		filter += filter_rows * filter_cols * count;
	}
	const int outcnt = out_rows * out_cols * batch;
	out += (blockIdx.x * out_cols + blockIdx.y) * batch;
	#pragma unroll
	for (i = 0; i < filter_per_thread; i++)
	{
		const float bias = shared_bias[i + threadIdx.y * filter_per_thread];
		#pragma unroll
		for (j = 0; j < input_per_thread; j++)
			out[(i + threadIdx.y * filter_per_thread) * outcnt + j + threadIdx.x * input_per_thread] = max(0.0, prod[i][j] + bias);
	}
}

template <int channel_per_thread, int filter_per_thread, int batch_per_thread>
__global__ void _cog_kern_convolutional_backward_propagate_delta(const int strides, const int border, const int batch,
		float* input, const int rows, const int cols, const int channels,
		float* out, const int out_rows, const int out_cols,
		float* out_grad, float* delta,
		float* filter, const int filter_rows, const int filter_cols, const int count)
{
	// gridDim.x == filter_rows
	// gridDim.y == filter_cols
	assert(gridDim.z == batch / batch_per_thread);
	extern __shared__ float shared[];
	float* shared_block = &shared[0];
	float* shared_out = &shared[batch_per_thread * channels];
	float* shared_grad = &shared[batch_per_thread * (channels + count)];
	float prod[channel_per_thread][filter_per_thread];
	// channel_per_thread * blockDim.x == channels
	// filter_per_thread * blockDim.y == filter_count
	assert(channel_per_thread * blockDim.x == channels);
	assert(filter_per_thread * blockDim.y == count);
	const int thidx = threadIdx.x + threadIdx.y * blockDim.x;
	const int thcnt = blockDim.x * blockDim.y;
	assert(batch % batch_per_thread == 0);
	assert(thcnt % batch_per_thread == 0);
	int i, j, k, x, y;
	#pragma unroll
	for (i = 0; i < channel_per_thread; i++)
		#pragma unroll
		for (j = 0; j < filter_per_thread; j++)
			prod[i][j] = 0;
	const int bxidx = thidx % batch_per_thread;
	const int byidx = thidx / batch_per_thread;
	const int incnt = rows * cols * batch;
	input += (blockIdx.x * cols + blockIdx.y) * batch + blockIdx.z * batch_per_thread + byidx * incnt + bxidx;
	const int outcnt = out_rows * out_cols * batch;
	out += blockIdx.z * batch_per_thread + byidx * outcnt + bxidx;
	out_grad += blockIdx.z * batch_per_thread + byidx * outcnt + bxidx;
	const int block_loads = (batch_per_thread * channels + thcnt - 1) / thcnt;
	const int out_loads = (batch_per_thread * count + thcnt - 1) / thcnt;
	const int block_loads_factor = (thcnt / batch_per_thread) * incnt;
	const int out_loads_factor = (thcnt / batch_per_thread) * outcnt;
	const int filter_idx = threadIdx.y * filter_per_thread;
	const int channel_idx = threadIdx.x * channel_per_thread;
	for (y = 0; y < out_rows; y++)
	{
		const int iy = blockIdx.x + y * strides - border;
		for (x = 0; x < out_cols; x++)
		{
			const int ix = blockIdx.y + x * strides - border;
			if (iy >= 0 && iy < rows && ix >= 0 && ix < cols)
			{
				#pragma unroll
				for (i = 0; i < block_loads; i++)
					if (thidx + i * thcnt < batch_per_thread * channels)
						shared_block[thidx + i * thcnt] = input[((y * strides - border) * cols + x * strides - border) * batch + i * block_loads_factor];
				#pragma unroll
				for (i = 0; i < out_loads; i++)
					if (thidx + i * thcnt < batch_per_thread * count)
						shared_out[thidx + i * thcnt] = out[x * batch + i * out_loads_factor],
						shared_grad[thidx + i * thcnt] = out_grad[x * batch + i * out_loads_factor];
				__syncthreads();
				#pragma unroll
				for (k = 0; k < batch_per_thread; k++)
					#pragma unroll
					for (i = 0; i < filter_per_thread; i++)
						if (shared_out[(i + filter_idx) * batch_per_thread + k] > 0)
							#pragma unroll
							for (j = 0; j < channel_per_thread; j++)
								prod[j][i] += shared_block[(j + channel_idx) * batch_per_thread + k] * shared_grad[(i + filter_idx) * batch_per_thread + k];
				__syncthreads();
			}
		}
		out += out_cols * batch;
		out_grad += out_cols * batch;
	}
	delta += (blockIdx.x * filter_cols + blockIdx.y) * count + blockIdx.z * filter_rows * filter_cols * count * channels;
	const int deltacnt = filter_rows * filter_cols * count;
	#pragma unroll
	for (i = 0; i < channel_per_thread; i++)
		#pragma unroll
		for (j = 0; j < filter_per_thread; j++)
			delta[(i + channel_idx) * deltacnt + j + filter_idx] = prod[i][j];
}

#include <sys/time.h>
#include <ctype.h>

static unsigned int get_current_time(void)
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000 + tv.tv_usec / 1000;
}

static void _cog_convolutional_forward_propagate(ccv_convnet_layer_t* layer, int batch, int rows, int cols, int ch, float* a, float* d, float** b)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int out_rows, out_cols;
	_ccv_convnet_compute_output_scale(rows, cols, layer, &out_rows, &out_cols);
	assert(b);
	float* db = *b;
	if (!db)
		hipMalloc(&db, sizeof(float) * out_rows * out_cols * layer->net.convolutional.count * batch);
	*b = db;
	dim3 threads_per_block(batch / 8, layer->net.convolutional.count / 4);
	dim3 num_blocks(out_rows, out_cols);
	int shared_memory_size = sizeof(float) * (batch + layer->net.convolutional.count);
	_cog_kern_convolutional_forward_propagate
		<8, 4>
		<<<num_blocks, threads_per_block, shared_memory_size + /* need extra space for bias */ sizeof(float) * layer->net.convolutional.count>>>
		(layer->net.convolutional.strides, layer->net.convolutional.border, batch,
		 a, rows, cols, ch,
		 db, out_rows, out_cols,
		 layer->w, layer->net.convolutional.rows, layer->net.convolutional.cols, layer->net.convolutional.count,
		 layer->bias);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("cuda elapsed time forward propagate: %.2lf\n", elapsed_time);
}

static void _ccv_convnet_convolutional_forward_propagate(ccv_convnet_layer_t* layer, ccv_dense_matrix_t* a, ccv_dense_matrix_t* d, ccv_dense_matrix_t** b)
{
	int rows, cols;
	_ccv_convnet_compute_output_scale(a->rows, a->cols, layer, &rows, &cols);
	int ch = layer->net.convolutional.channels;
	int count = layer->net.convolutional.count;
	int strides = layer->net.convolutional.strides;
	int border = layer->net.convolutional.border;
	int kernel_rows = layer->net.convolutional.rows;
	int kernel_cols = layer->net.convolutional.cols;
	int type = CCV_32F | count;
	assert(CCV_GET_CHANNEL(a->type) == ch);
	assert(CCV_GET_DATA_TYPE(a->type) == CCV_32F);
	ccv_dense_matrix_t* db = *b = ccv_dense_matrix_renew(*b, rows, cols, type, type, 0);
	int i, j, x, y, k;
#define for_block(act_block_setup, act_block_begin, act_block_end) \
	for (k = 0; k < count; k++) \
	{ \
		float* ap = a->data.f32; \
		float* bp = db->data.f32 + k; \
		float* layer_w = layer->w + k * kernel_rows * kernel_cols * ch; \
		float bias = layer->bias[k]; \
		act_block_setup; \
		for (i = 0; i < db->rows; i++) \
		{ \
			int comy = ccv_max(i * strides - border, 0) - (i * strides - border); \
			int maxy = kernel_rows - comy - (i * strides + kernel_rows - ccv_min(a->rows + border, i * strides + kernel_rows)); \
			comy *= ch * kernel_cols; \
			for (j = 0; j < db->cols; j++) \
			{ \
				act_block_begin; \
				float v = bias; \
				int comx = (ccv_max(j * strides - border, 0) - (j * strides - border)) * ch; \
				int maxx = kernel_cols * ch - comx - (j * strides + kernel_cols - ccv_min(a->cols + border, j * strides + kernel_cols)) * ch; \
				float* w = layer_w + comx + comy; \
				float* apz = ap + ccv_max(j * strides - border, 0) * ch; \
				/* when we have border, we simply do zero padding */ \
				for (y = 0; y < maxy; y++) \
				{ \
					for (x = 0; x < maxx; x++) \
						v += w[x] * apz[x]; \
					w += kernel_cols * ch; \
					apz += a->cols * ch; \
				} \
				bp[j * count] = ccv_max(0, v) /* ReLU */; \
				act_block_end; \
			} \
			bp += db->cols * count; \
			ap += a->cols * ch * (ccv_max((i + 1) * strides - border, 0) - ccv_max(i * strides - border, 0)); \
		} \
	}
	if (d)
	{
#define act_block_setup \
		int* dp = d->data.i32 + k;
#define act_block_begin \
		if (!*dp) \
		{
#define act_block_end \
		} else \
			bp[j * count] = 0; \
		dp += count;
		for_block(act_block_setup, act_block_begin, act_block_end);
#undef act_block_setup
#undef act_block_begin
#undef act_block_end
	} else {
		for_block(/* empty act block setup */, /* empty act block begin */, /* empty act block end */);
	}
#undef for_block
}

static void _cog_convnet_convolutional_backward_propagate(ccv_convnet_layer_t* layer, int batch, int rows, int cols, int ch, float* a, float* n, float* d, float* m, float** b, ccv_convnet_layer_t* update)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	assert(layer->net.convolutional.count % 4 == 0);
	int out_rows, out_cols;
	_ccv_convnet_compute_output_scale(rows, cols, layer, &out_rows, &out_cols);
	dim3 threads_per_block(ch, layer->net.convolutional.count);
	assert(batch % 8 == 0);
	dim3 num_blocks(layer->net.convolutional.rows, layer->net.convolutional.cols, batch / 8);
	int shared_memory_size = sizeof(float) * (8 * (ch + layer->net.convolutional.count * 2));
	_cog_kern_convolutional_backward_propagate_delta
	<1, 1, 8>
	<<<num_blocks, threads_per_block, shared_memory_size>>>
	(layer->net.convolutional.strides, layer->net.convolutional.border, batch,
		m, rows, cols, ch,
		n, out_rows, out_cols,
		a, update->w,
		layer->w, layer->net.convolutional.rows, layer->net.convolutional.cols, layer->net.convolutional.count);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("cuda elapsed time of backward propagate: %.2lf\n", elapsed_time);
}

// compute back propagated gradient & weight update delta
static void _ccv_convnet_convolutional_backward_propagate(ccv_convnet_layer_t* layer, ccv_dense_matrix_t* a, ccv_dense_matrix_t* n, ccv_dense_matrix_t* d, ccv_dense_matrix_t* m, ccv_dense_matrix_t** b, ccv_convnet_layer_t* update_params)
{
	// a is the input gradient (for back prop), d is the dropout,
	// x is the input (for forward prop), b is the output gradient (gradient, or known as propagated error)
	// note that y (the output from forward prop) is not included because the full connect net is simple enough that we don't need it
	int rows, cols;
	_ccv_convnet_compute_output_scale(m->rows, m->cols, layer, &rows, &cols);
	int ch = layer->net.convolutional.channels;
	int count = layer->net.convolutional.count;
	int strides = layer->net.convolutional.strides;
	int border = layer->net.convolutional.border;
	int kernel_rows = layer->net.convolutional.rows;
	int kernel_cols = layer->net.convolutional.cols;
	assert(a->rows == rows);
	assert(a->cols == cols);
	assert(CCV_GET_CHANNEL(a->type) == count);
	int a_rows = a->rows, a_cols = a->cols, a_ch = CCV_GET_CHANNEL(a->type);
	a->rows = rows, a->cols = cols, a->type = (a->type - a_ch) | count;
	assert(CCV_GET_CHANNEL(m->type) == ch);
	assert(CCV_GET_DATA_TYPE(m->type) == CCV_32F);
	int i, j, x, y, k;
	// update weight gradient
#define for_block_w(act_block_setup, act_block_begin, act_block_end) \
	for (k = 0; k < count; k++) \
	{ \
		float* mp = m->data.f32; \
		float* ap = a->data.f32 + k; \
		float* np = n->data.f32 + k; \
		float* update_w = update_params->w + k * kernel_rows * kernel_cols * ch; \
		float bias = 0; \
		act_block_setup; \
		for (i = 0; i < rows; i++) \
		{ \
			int comy = ccv_max(i * strides - border, 0) - (i * strides - border); \
			int maxy = kernel_rows - comy - (i * strides + kernel_rows - ccv_min(m->rows + border, i * strides + kernel_rows)); \
			comy *= ch * kernel_cols; \
			for (j = 0; j < cols; j++) \
			{ \
				act_block_begin; \
				if (np[j * count] > 0) \
				{ /* when np is bigger than 0, relu continues to update the weight, otherwise it stops */ \
					float v = ap[j * count]; \
					bias += v; \
					int comx = (ccv_max(j * strides - border, 0) - (j * strides - border)) * ch; \
					int maxx = kernel_cols * ch - comx - (j * strides + kernel_cols - ccv_min(m->cols + border, j * strides + kernel_cols)) * ch; \
					float* w = update_w + comx + comy; \
					float* mpz = mp + ccv_max(j * strides - border, 0) * ch; \
					/* when we have border, we simply do zero padding */ \
					for (y = 0; y < maxy; y++) \
					{ \
						for (x = 0; x < maxx; x++) \
							w[x] += v * mpz[x]; \
						w += kernel_cols * ch; \
						mpz += m->cols * ch; \
					} \
				} \
				act_block_end; \
			} \
			ap += a->cols * count; \
			np += n->cols * count; \
			mp += m->cols * ch * (ccv_max((i + 1) * strides - border, 0) - ccv_max(i * strides - border, 0)); \
		} \
		update_params->bias[k] = bias; \
	}
	ccv_dense_matrix_t* db = 0;
	if (b)
	{
		db = *b = ccv_dense_matrix_renew(*b, m->rows, m->cols, CCV_32F | CCV_GET_CHANNEL(m->type), CCV_32F | CCV_GET_CHANNEL(m->type), 0);
		// clear it up before propagate result
		ccv_zero(db);
	}
#define for_block_b(act_block_setup, act_block_begin, act_block_end) \
	for (k = 0; k < count; k++) \
	{ \
		float* bp = db->data.f32; \
		float* ap = a->data.f32 + k; \
		float* np = n->data.f32 + k; \
		float* layer_w = layer->w + k * kernel_rows * kernel_cols * ch; \
		act_block_setup; \
		for (i = 0; i < rows; i++) \
		{ \
			int comy = ccv_max(i * strides - border, 0) - (i * strides - border); \
			int maxy = kernel_rows - comy - (i * strides + kernel_rows - ccv_min(db->rows + border, i * strides + kernel_rows)); \
			comy *= ch * kernel_cols; \
			for (j = 0; j < cols; j++) \
			{ \
				act_block_begin; \
				if (np[j * count] > 0) \
				{ /* when np is bigger than 0, relu continues to update the weight, otherwise it stops */ \
					float v = ap[j * count]; \
					int comx = (ccv_max(j * strides - border, 0) - (j * strides - border)) * ch; \
					int maxx = kernel_cols * ch - comx - (j * strides + kernel_cols - ccv_min(db->cols + border, j * strides + kernel_cols)) * ch; \
					float* w = layer_w + comx + comy; \
					float* bpz = bp + ccv_max(j * strides - border, 0) * ch; \
					/* when we have border, we simply do zero padding */ \
					for (y = 0; y < maxy; y++) \
					{ \
						for (x = 0; x < maxx; x++) \
							bpz[x] += v * w[x]; \
						w += kernel_cols * ch; \
						bpz += db->cols * ch; \
					} \
				} \
				act_block_end; \
			} \
			ap += a->cols * count; \
			np += n->cols * count; \
			bp += db->cols * ch * (ccv_max((i + 1) * strides - border, 0) - ccv_max(i * strides - border, 0)); \
		} \
	}
	if (d)
	{
#define act_block_setup \
		int* dp = d->data.i32 + k;
#define act_block_begin \
		if (!*dp) \
		{
#define act_block_end \
		} \
		dp += count;
		for_block_w(act_block_setup, act_block_begin, act_block_end);
		if (db)
			for_block_b(act_block_setup, act_block_begin, act_block_end);
#undef act_block_setup
#undef act_block_begin
#undef act_block_end
	} else {
		for_block_w(/* empty act block setup */, /* empty act block begin */, /* empty act block end */);
		if (db)
			for_block_b(/* empty act block setup */, /* empty act block begin */, /* empty act block end */);
	}
#undef for_block_w
#undef for_block_b
	a->rows = a_rows, a->cols = a_cols, a->type = (a->type - CCV_GET_CHANNEL(a->type)) | a_ch;
}

static void _cog_convnet_reserve_on_device(ccv_convnet_t* convnet)
{
	assert(GPU(convnet) == 0);
	convnet->reserved = (cog_convnet_t*)ccmalloc(sizeof(cog_convnet_t) + sizeof(ccv_convnet_layer_t) * convnet->count * 2);
	GPU(convnet)->layers = (ccv_convnet_layer_t*)(GPU(convnet) + 1);
	GPU(convnet)->updates = GPU(convnet)->layers + convnet->count;
	memcpy(GPU(convnet)->layers, convnet->layers, sizeof(ccv_convnet_layer_t) * convnet->count);
	memcpy(GPU(convnet)->updates, convnet->layers, sizeof(ccv_convnet_layer_t) * convnet->count);
	ccv_convnet_layer_t* layers = GPU(convnet)->layers;
	ccv_convnet_layer_t* updates = GPU(convnet)->updates;
	int i;
	for (i = 0; i < convnet->count; i++)
		switch (layers[i].type)
		{
			case CCV_CONVNET_CONVOLUTIONAL:
				assert(updates[i].type == CCV_CONVNET_CONVOLUTIONAL);
				layers[i].w = 0;
				hipMalloc(&layers[i].w, sizeof(float) * (layers[i].wnum + layers[i].net.convolutional.count));
				assert(layers[i].w);
				layers[i].bias = layers[i].w + layers[i].wnum;
				// this is wrong, I need to rewind w
				hipMemcpy(layers[i].w, convnet->layers[i].w, sizeof(float) * (layers[i].wnum + layers[i].net.convolutional.count), hipMemcpyHostToDevice);
				updates[i].w = 0;
				hipMalloc(&updates[i].w, sizeof(float) * (updates[i].wnum * 16 + updates[i].net.convolutional.count));
				assert(updates[i].w);
				updates[i].bias = updates[i].w + updates[i].wnum;
				break;
			case CCV_CONVNET_FULL_CONNECT:
				assert(updates[i].type == CCV_CONVNET_FULL_CONNECT);
				layers[i].w = 0;
				hipMalloc(&layers[i].w, sizeof(float) * (layers[i].wnum + layers[i].net.full_connect.count));
				assert(layers[i].w);
				layers[i].bias = layers[i].w + layers[i].wnum;
				hipMemcpy(layers[i].w, convnet->layers[i].w, sizeof(float) * (layers[i].wnum + layers[i].net.full_connect.count), hipMemcpyHostToDevice);
				updates[i].w = 0;
				hipMalloc(&updates[i].w, sizeof(float) * (updates[i].wnum * 16 + updates[i].net.full_connect.count));
				updates[i].bias = updates[i].w + updates[i].wnum;
				break;
			case CCV_CONVNET_MAX_POOL:
			case CCV_CONVNET_AVERAGE_POOL:
				assert(updates[i].type == layers[i].type);
				updates[i].w = updates[i].bias = 0;
				layers[i].w = layers[i].bias = 0;
				break;
		}
}

void cog_convnet_encode(ccv_convnet_t* convnet, ccv_dense_matrix_t** a, ccv_dense_matrix_t** b, int batch)
{
	int ch = CCV_GET_CHANNEL(a[0]->type);
	int rows = a[0]->rows, cols = a[0]->cols;
	float* vec = 0;
	hipHostMalloc(&vec, sizeof(float) * batch * rows * cols * ch);
	int i, j, k, c, z;
	for (i = 0; i < batch; i++)
		for (k = 0; k < ch; k++)
			for (j = 0; j < rows * cols; j++)
				vec[i + (k * rows * cols + j) * batch] = a[i]->data.f32[j * ch + k];
	float* od_vec = 0;
	hipMalloc(&od_vec, sizeof(float) * batch * rows * cols * ch);
	hipMemcpy(od_vec, vec, sizeof(float) * batch * rows * cols * ch, hipMemcpyHostToDevice);
	float* od_out = 0;
	_cog_convolutional_forward_propagate(GPU(convnet)->layers, batch, rows, cols, ch, od_vec, 0, &od_out);
	_cog_convnet_convolutional_backward_propagate(GPU(convnet)->layers, batch, rows, cols, ch, od_out, od_out, 0, od_vec, 0, GPU(convnet)->updates);
	float* out_weights = 0;
	hipHostMalloc(&out_weights, sizeof(float) * convnet->layers->wnum * 16);
	assert(out_weights);
	hipMemcpy(out_weights, GPU(convnet)->updates->w, sizeof(float) * convnet->layers->wnum * 16, hipMemcpyDeviceToHost);
	int out_rows, out_cols;
	_ccv_convnet_compute_output_scale(rows, cols, convnet->layers, &out_rows, &out_cols);
	ccv_convnet_layer_t updates;
	updates.w = (float*)ccmalloc(sizeof(float) * (convnet->layers->wnum + convnet->layers->net.convolutional.count));
	memset(updates.w, 0, sizeof(float) * (convnet->layers->wnum + convnet->layers->net.convolutional.count));
	updates.bias = updates.w + convnet->layers->wnum;
	unsigned int elapsed_time = get_current_time();
	for (i = 0; i < batch; i++)
	{
		ccv_dense_matrix_t* b = 0;
		_ccv_convnet_convolutional_forward_propagate(convnet->layers, a[i], 0, &b);
		ccv_dense_matrix_t* out = 0;
		_ccv_convnet_convolutional_backward_propagate(convnet->layers, b, b, 0, a[i], &out, &updates);
		ccv_matrix_free(b);
		ccv_matrix_free(out);
	}
	elapsed_time = get_current_time() - elapsed_time;
	printf("cpu elapsed time of backward propagate: %u\n", elapsed_time);
	int filter_rows = convnet->layers->net.convolutional.rows;
	int filter_cols = convnet->layers->net.convolutional.cols;
	int filter_count = convnet->layers->net.convolutional.count;
	for (i = 0; i < filter_rows; i++)
		for (j = 0; j < filter_cols; j++)
			for (k = 0; k < filter_count; k++)
				for (c = 0; c < ch; c++)
				{
					float w = updates.w[(i * filter_cols + j) * ch + k * filter_cols * filter_rows * ch + c];
					float ow = out_weights[(i * filter_cols + j) * filter_count + k + c * filter_cols * filter_rows * filter_count];
					for (z = 1; z < 16; z++)
						ow += out_weights[z * filter_rows * filter_cols * filter_count * ch + (i * filter_cols + j) * filter_count + k + c * filter_cols * filter_rows * filter_count];
					float delta = fabsf(ow - w) / w;
					if (delta > 0.0001)
						printf("%d,%d,%d,%d: %f, %f\n", i, j, k, c, w, ow);
				}
}

void cog_convnet_classify(ccv_convnet_t* convnet, ccv_dense_matrix_t** a, int* labels, int batch)
{
}

void cog_convnet_supervised_train(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_array_t* tests, ccv_convnet_train_param_t params)
{
	assert(categorizeds->rnum >= 128);
	if (!GPU(convnet))
		_cog_convnet_reserve_on_device(convnet);
	int i;
	ccv_dense_matrix_t* a[128];
	for (i = 0; i < 128; i++)
	{
		ccv_categorized_t* categorized = (ccv_categorized_t*)ccv_array_get(categorizeds, i);
		ccv_dense_matrix_t* image = 0;
		ccv_read(categorized->file.filename, &image, CCV_IO_ANY_FILE | CCV_IO_RGB_COLOR);
		ccv_dense_matrix_t* b = 0;
		if (image->rows > 251 && image->cols > 251)
			ccv_resample(image, &b, 0, ccv_max(251, (int)(image->rows * 251.0 / image->cols + 0.5)), ccv_max(251, (int)(image->cols * 251.0 / image->rows + 0.5)), CCV_INTER_AREA);
		else if (image->rows < 251 || image->cols < 251)
			ccv_resample(image, &b, 0, ccv_max(251, (int)(image->rows * 251.0 / image->cols + 0.5)), ccv_max(251, (int)(image->cols * 251.0 / image->rows + 0.5)), CCV_INTER_CUBIC);
		else
			b = image;
		if (b != image)
			ccv_matrix_free(image);
		ccv_dense_matrix_t* c = 0;
		ccv_slice(b, (ccv_matrix_t**)&c, CCV_32F, 0, 0, 225, 225);
		int j, ch = CCV_GET_CHANNEL(c->type);
		for (j = 0; j < c->rows * c->cols * ch; j++)
			c->data.f32[j] = c->data.f32[j] / 255.0 * 2 - 1;
		a[i] = c;
		ccv_matrix_free(b);
	}
	cog_convnet_encode(convnet, a, 0, 128);
}

void cog_convnet_free(ccv_convnet_t* convnet)
{
	int i;
	ccv_convnet_layer_t* layers = GPU(convnet)->layers;
	for (i = 0; i < convnet->count; i++)
		hipFree(layers[i].w);
	ccfree(convnet);
}
