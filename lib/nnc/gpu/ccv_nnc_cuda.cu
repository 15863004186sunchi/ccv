extern "C" {
#include "ccv_nnc_cuda.h"
}
#include <hip/hip_runtime.h>

void* gcmalloc(size_t size)
{
	void* ptr = 0;
	hipMalloc(&ptr, size);
	return ptr;
}

void gcfree(void* ptr)
{
	hipFree(ptr);
}
