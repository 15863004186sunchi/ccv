extern "C" {
#include "ccv_nnc_cuda.h"
}
#include <hip/hip_runtime.h>

void* gcmalloc(size_t size)
{
	void* ptr = 0;
	hipMalloc(&ptr, size);
	return ptr;
}

void gcmemcpy(void* dst, const void* src, size_t size, int kind)
{
	if (kind == CCV_NNC_MEMCPY_CPU_TO_GPU)
		hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
	else if (kind == CCV_NNC_MEMCPY_GPU_TO_CPU)
		hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void gcfree(void* ptr)
{
	hipFree(ptr);
}
