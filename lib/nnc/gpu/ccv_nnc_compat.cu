#include "ccv_nnc_compat.h"
extern "C" {
#include <nnc/ccv_nnc_easy.h>
}

void* cumalloc(int device, size_t size)
{
	void* ptr = 0;
	hipSetDevice(device);
	hipMalloc(&ptr, size);
	return ptr;
}

void cufree(int device, void* ptr)
{
	hipSetDevice(device);
	hipFree(ptr);
}

typedef struct {
	int type; // Kept the type specifier.
	hipStream_t stream;
	hipblasHandle_t cublas;
#ifdef HAVE_CUDNN
	hipdnnHandle_t cudnn;
#endif
} ccv_nnc_stream_context_compat_t;

ccv_nnc_stream_context_t* ccv_nnc_init_stream_context(ccv_nnc_stream_context_t* stream_context)
{
	assert(CCV_STREAM_GET_CONTEXT(((int*)stream_context)[0]) == CCV_STREAM_CONTEXT_GPU);
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)ccrealloc(stream_context, sizeof(ccv_nnc_stream_context_compat_t));
	int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
	hipSetDevice(device);
	hipStreamCreate(&stream_compat->stream);
	stream_compat->cublas = 0;
#ifdef HAVE_CUDNN
	stream_compat->cudnn = 0;
#endif
	return (ccv_nnc_stream_context_t*)stream_compat;
}

void ccv_nnc_synchronize_stream_context(const ccv_nnc_stream_context_t* stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
	hipSetDevice(device);
	hipStreamSynchronize(stream_compat->stream);
}

void ccv_nnc_deinit_stream_context(ccv_nnc_stream_context_t* stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
	hipSetDevice(device);
	hipStreamDestroy(stream_compat->stream);
	if (stream_compat->cublas)
		hipblasDestroy(stream_compat->cublas);
#ifdef HAVE_CUDNN
	if (stream_compat->cudnn)
		hipdnnDestroy(stream_compat->cudnn);
#endif
}

int ccv_nnc_stream_context_get_device(const ccv_nnc_stream_context_t* stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	return CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
}

hipStream_t ccv_nnc_stream_context_get_stream(const ccv_nnc_stream_context_t* stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	return stream_compat->stream;
}

hipblasHandle_t ccv_nnc_stream_context_get_cublas(const ccv_nnc_stream_context_t* stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat->cublas)
	{
		int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
		hipSetDevice(device);
		hipblasCreate(&stream_compat->cublas);
		hipblasSetStream(stream_compat->cublas, stream_compat->stream);
	}
	return stream_compat->cublas;
}

#ifdef HAVE_CUDNN
hipdnnHandle_t ccv_nnc_stream_context_get_cudnn(const ccv_nnc_stream_context_t* stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat->cudnn)
	{
		int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
		hipSetDevice(device);
		hipdnnCreate(&stream_compat->cudnn);
		hipdnnSetStream(stream_compat->cudnn, stream_compat->stream);
	}
	return stream_compat->cudnn;
}

hipdnnConvolutionDescriptor_t ccv_nnc_stream_context_get_convolution_descriptor(const ccv_nnc_stream_context_t* stream_context)
{
	hipdnnConvolutionDescriptor_t desc;
	hipdnnCreateConvolutionDescriptor(&desc);
	return desc;
}

hipdnnTensorDescriptor_t ccv_nnc_stream_context_get_tensor_descriptor(const ccv_nnc_stream_context_t* stream_context)
{
	hipdnnTensorDescriptor_t desc;
	hipdnnCreateTensorDescriptor(&desc);
	return desc;
}

hipdnnFilterDescriptor_t ccv_nnc_stream_context_get_filter_descriptor(const ccv_nnc_stream_context_t* stream_context)
{
	hipdnnFilterDescriptor_t desc;
	hipdnnCreateFilterDescriptor(&desc);
	return desc;
}

void ccv_nnc_stream_context_return_convolution_descriptor(const ccv_nnc_stream_context_t* stream_context, hipdnnConvolutionDescriptor_t convolution_descriptor)
{
	hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
}

void ccv_nnc_stream_context_return_tensor_descriptor(const ccv_nnc_stream_context_t* stream_context, hipdnnTensorDescriptor_t tensor_descriptor)
{
	hipdnnDestroyTensorDescriptor(tensor_descriptor);
}

void ccv_nnc_stream_context_return_filter_descriptor(const ccv_nnc_stream_context_t* stream_context, hipdnnFilterDescriptor_t filter_descriptor)
{
	hipdnnDestroyFilterDescriptor(filter_descriptor);
}

ccv_nnc_cudnn_tensor_view_descriptor_t ccv_nnc_cudnn_get_tensor_view_descriptor(const ccv_nnc_stream_context_t* stream_context, const ccv_nnc_tensor_view_t* tensor)
{
	ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc = {
		stream_context,
		ccv_nnc_stream_context_get_tensor_descriptor(stream_context),
		tensor->data,
	};
	// N is the outer one nevertheless.
	assert(tensor->info.format == CCV_TENSOR_FORMAT_NCHW || tensor->info.format == CCV_TENSOR_FORMAT_NHWC);
	// Fill up dimensions with 1s.
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {0};
	int i;
	const int nd = CCV_NNC_MAX_DIM + 2;
	// This has to follow NCHW
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
		for (i = 0; i < nd; i++)
			dim[i] = ccv_max(1, tensor->info.dim[nd - 1 - i]);
	else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		dim[0] = ccv_max(1, tensor->info.dim[nd - 1]);
		dim[1] = ccv_max(1, tensor->info.dim[0]);
		for (i = 2; i < nd; i++)
			dim[i] = ccv_max(1, tensor->info.dim[nd - i]);
	}
	const int* inc = CCV_IS_TENSOR_VIEW(tensor) ? tensor->inc : tensor->info.dim;
	int stride[CCV_NNC_MAX_DIM_ALLOC];
	stride[nd - 1] = 1;
	// Compute the stride from inc, so it will fit the tensor view.
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
		for (i = 1; i < CCV_NNC_MAX_DIM_ALLOC && dim[i] > 0; i++)
			stride[nd - 1 - i] = stride[nd - i] * ccv_max(1, inc[i - 1]);
	else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		stride[1] = 1;
		stride[nd - 1] = ccv_max(1, inc[0]);
		for (i = 0; i < nd - 3; i++)
			stride[nd - 2 - i] = stride[nd - 1 - i] * ccv_max(1, inc[i + 1]);
		stride[0] = stride[2] * ccv_max(1, inc[nd - 2]);
	}
	assert_cudnn(hipdnnSetTensorNdDescriptor(tensor_desc.descriptor, HIPDNN_DATA_FLOAT, nd, dim, stride));
	return tensor_desc;
}

void ccv_nnc_cudnn_deinit_tensor_view_descriptor(const ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc)
{
	ccv_nnc_stream_context_return_tensor_descriptor(tensor_desc.stream_context, tensor_desc.descriptor);
}

ccv_nnc_cudnn_filter_descriptor_t ccv_nnc_cudnn_get_filter_descriptor(const ccv_nnc_stream_context_t* stream_context, const ccv_nnc_tensor_t* tensor)
{
	ccv_nnc_cudnn_filter_descriptor_t filter_desc = {
		stream_context,
		ccv_nnc_stream_context_get_filter_descriptor(stream_context),
		tensor->data,
	};
	assert(!CCV_IS_TENSOR_VIEW(tensor));
	int nd = ccv_nnc_tensor_nd(tensor->info);
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {0};
	int i;
	// Reorder since nnc have different idea about NCHW and NHWC (N is in 3, C is in 0).
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		for (i = 0; i < nd; i++)
			dim[i] = tensor->info.dim[nd - 1 - i];
		assert_cudnn(hipdnnSetFilterNdDescriptor(filter_desc.descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, nd, dim));
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		dim[0] = tensor->info.dim[nd - 1];
		dim[1] = tensor->info.dim[0];
		for (i = 2; i < nd; i++)
			dim[i] = tensor->info.dim[nd - i];
		assert_cudnn(hipdnnSetFilterNdDescriptor(filter_desc.descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NHWC, nd, dim));
	}
	return filter_desc;
}

void ccv_nnc_cudnn_deinit_filter_descriptor(const ccv_nnc_cudnn_filter_descriptor_t filter_desc)
{
	ccv_nnc_stream_context_return_filter_descriptor(filter_desc.stream_context, filter_desc.descriptor);
}

ccv_nnc_cudnn_convolution_descriptor_t ccv_nnc_cudnn_get_convolution_descriptor(const ccv_nnc_stream_context_t* stream_context, const ccv_nnc_hint_t hint)
{
	ccv_nnc_cudnn_convolution_descriptor_t convolution_desc = {
		stream_context,
		ccv_nnc_stream_context_get_convolution_descriptor(stream_context),
	};
	int i;
	int p[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		p[i] = ccv_max(hint.border.begin[CCV_NNC_MAX_DIM - i], hint.border.end[CCV_NNC_MAX_DIM - i]);
	int v[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		v[i] = hint.stride.dim[CCV_NNC_MAX_DIM - i];
	int u[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		u[i] = 1;
	assert_cudnn(hipdnnSetConvolutionNdDescriptor(convolution_desc.descriptor, CCV_NNC_MAX_DIM, p, v, u, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	return convolution_desc;
}

void ccv_nnc_cudnn_deinit_convolution_descriptor(const ccv_nnc_cudnn_convolution_descriptor_t convolution_desc)
{
	ccv_nnc_stream_context_return_convolution_descriptor(convolution_desc.stream_context, convolution_desc.descriptor);
}
#endif

static void _ccv_nnc_cufree_stream_callback(hipStream_t stream, hipError_t status, void* ptr)
{
	hipFree(ptr);
}

void hipFreeAsync(void* ptr, hipStream_t stream)
{
	hipStreamAddCallback(stream, _ccv_nnc_cufree_stream_callback, ptr, 0);
}
