#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>

// The header for CUDA related objects.
#include "ccv_nnc_cmd.h"

// nvcc is a C++ compiler, need to specify this is a "C" function to avoid name mangling.
extern "C" void ccv_nnc_gpu_cudnn_init(ccv_nnc_cmd_api_t cmd_api[]);

#ifdef HAVE_CUDNN
static int _ccv_nnc_conv_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* inputs, const int input_size, ccv_nnc_tensor_t** outputs, const int output_size, const ccv_nnc_stream_context_t* stream_context)
{
	assert(input_size == 3);
	assert(output_size == 1);
	assert(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	hipdnnTensorDescriptor_t a_desc, bias_desc, b_desc;
	hipdnnFilterDescriptor_t w_desc;
	hipdnnCreateTensorDescriptor(&a_desc);
	hipdnnCreateFilterDescriptor(&w_desc);
	hipdnnCreateTensorDescriptor(&bias_desc);
	hipdnnCreateTensorDescriptor(&b_desc);

	hipdnnDestroyTensorDescriptor(a_desc);
	hipdnnDestroyFilterDescriptor(w_desc);
	hipdnnDestroyTensorDescriptor(bias_desc);
	hipdnnDestroyTensorDescriptor(b_desc);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_conv_forw_autotune(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* inputs, const int input_size, ccv_nnc_tensor_t** outputs, const int output_size, const ccv_nnc_stream_context_t* stream_context)
{
	return 0; // Return the most efficient algorithm, return -1 if cannot find one.
}
#endif

//@ccv_nnc_init CCV_NNC_BACKEND_GPU_CUDNN
void ccv_nnc_gpu_cudnn_init(ccv_nnc_cmd_api_t cmd_api[])
{
#ifdef HAVE_CUDNN
	/* Convolutional layer */
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].tensor_formats = CCV_TENSOR_FORMAT_NHWC;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].tensor_memory = CCV_TENSOR_GPU_MEMORY;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].algorithms = 0;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].exec = _ccv_nnc_conv_forw;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].autotune = _ccv_nnc_conv_forw_autotune;
	/* Full connect layer */
	/* Max pool layer */
	/* Average pool layer */
	/* Softmax layer */
	/* ReLU activation */
#endif
}
