#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>

// The header for CUDA related objects.
#include "ccv_nnc_cmd.h"

// nvcc is a C++ compiler, need to specify this is a "C" function to avoid name mangling.
extern "C" void ccv_nnc_gpu_cudnn_init(ccv_nnc_cmd_api_t cmd_api[]);

#ifdef HAVE_CUDNN

#define checkCUDNN(status) {                                      \
	if (status != HIPDNN_STATUS_SUCCESS) {                         \
		printf("%s:%d\nCUDNN failure\nError: %s\n",               \
				__FILE__, __LINE__, hipdnnGetErrorString(status)); \
		hipDeviceReset();                                        \
		exit(EXIT_FAILURE);                                       \
	}                                                             \
}

static void _ccv_nnc_set_tensor_nd_desc(const ccv_nnc_tensor_view_t* tensor, hipdnnTensorDescriptor_t desc)
{
	// N is the outer one nevertheless.
	assert(tensor->info.format == CCV_TENSOR_FORMAT_NCHW || tensor->info.format == CCV_TENSOR_FORMAT_NHWC);
	// Fill up dimensions with 1s.
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {0};
	int i;
	const int nd = CCV_NNC_MAX_DIM + 2;
	// This has to follow NCHW
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
		for (i = 0; i < nd; i++)
			dim[i] = ccv_max(1, tensor->info.dim[nd - 1 - i]);
	else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		dim[0] = ccv_max(1, tensor->info.dim[nd - 1]);
		dim[1] = ccv_max(1, tensor->info.dim[0]);
		for (i = 2; i < nd; i++)
			dim[i] = ccv_max(1, tensor->info.dim[nd - i]);
	}
	printf("tensor %d %d %d %d\n", dim[0], dim[1], dim[2], dim[3]);
	const int* inc = CCV_IS_TENSOR_VIEW(tensor) ? tensor->inc : tensor->info.dim;
	int stride[CCV_NNC_MAX_DIM_ALLOC];
	stride[nd - 1] = 1;
	// Compute the stride from inc, so it will fit the tensor view.
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
		for (i = 1; i < CCV_NNC_MAX_DIM_ALLOC && dim[i] > 0; i++)
			stride[nd - 1 - i] = stride[nd - i] * ccv_max(1, inc[i - 1]);
	else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		stride[1] = 1;
		stride[nd - 1] = ccv_max(1, inc[0]);
		for (i = 0; i < nd - 3; i++)
			stride[nd - 2 - i] = stride[nd - 1 - i] * ccv_max(1, inc[i + 1]);
		stride[0] = stride[2] * ccv_max(1, inc[nd - 2]);
	}
	printf("tensor stride %d %d %d %d\n", stride[0], stride[1], stride[2], stride[3]);
	hipdnnSetTensorNdDescriptor(desc, HIPDNN_DATA_FLOAT, nd, dim, stride);
}

static void _ccv_nnc_set_filter_nd_desc(const ccv_nnc_tensor_t* tensor, hipdnnFilterDescriptor_t desc)
{
	assert(!CCV_IS_TENSOR_VIEW(tensor));
	int nd = ccv_nnc_tensor_nd(tensor->info);
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {0};
	int i;
	// Reorder since nnc have different idea about NCHW and NHWC (N is in 3, C is in 0).
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		for (i = 0; i < nd; i++)
			dim[i] = tensor->info.dim[nd - 1 - i];
		printf("filter %d %d %d %d\n", dim[0], dim[1], dim[2], dim[3]);
		hipdnnSetFilterNdDescriptor(desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, nd, dim);
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		dim[0] = tensor->info.dim[nd - 1];
		dim[1] = tensor->info.dim[0];
		for (i = 2; i < nd; i++)
			dim[i] = tensor->info.dim[nd - i];
		printf("filter %d %d %d %d\n", dim[0], dim[1], dim[2], dim[3]);
		hipdnnSetFilterNdDescriptor(desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, nd, dim);
	}
}

enum {
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_GEMM, // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_GEMM, // HIPDNN_CONVOLUTION_FWD_ALGO_GEMM
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_DIRECT, // HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT, // HIPDNN_CONVOLUTION_FWD_ALGO_FFT
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT_TILING, // HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT_WINOGRAD, // HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT
};

static int _ccv_nnc_conv_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* inputs, const int input_size, ccv_nnc_tensor_t** outputs, const int output_size, const ccv_nnc_stream_context_t* stream_context)
{
	assert(input_size == 3);
	assert(output_size == 1);
	assert(stream_context);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	hipdnnTensorDescriptor_t a_desc, bias_desc, b_desc;
	hipdnnFilterDescriptor_t w_desc;
	hipdnnCreateTensorDescriptor(&a_desc);
	const ccv_nnc_tensor_t* a = inputs[0];
	_ccv_nnc_set_tensor_nd_desc((const ccv_nnc_tensor_view_t*)a, a_desc);
	hipdnnCreateFilterDescriptor(&w_desc);
	const ccv_nnc_tensor_t* w = inputs[1];
	_ccv_nnc_set_filter_nd_desc(w, w_desc);
	hipdnnCreateTensorDescriptor(&bias_desc);
	const ccv_nnc_tensor_t* bias = inputs[2];
	_ccv_nnc_set_tensor_nd_desc((const ccv_nnc_tensor_view_t*)bias, bias_desc);
	hipdnnCreateTensorDescriptor(&b_desc);
	ccv_nnc_tensor_t* b = outputs[0];
	_ccv_nnc_set_tensor_nd_desc((const ccv_nnc_tensor_view_t*)b, b_desc);
	hipdnnConvolutionDescriptor_t conv_desc;
	hipdnnCreateConvolutionDescriptor(&conv_desc);
	int i;
	int p[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		p[i] = ccv_max(hint.border.begin[CCV_NNC_MAX_DIM - i], hint.border.end[CCV_NNC_MAX_DIM - i]);
	int v[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		v[i] = hint.stride.dim[CCV_NNC_MAX_DIM - i];
	int u[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		u[i] = 1;
	hipdnnSetConvolutionNdDescriptor(conv_desc, CCV_NNC_MAX_DIM, p, v, u, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

	hipdnnConvolutionFwdAlgo_t algo;
	// Choose an algorithm
	switch (cmd.algorithm)
	{
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_GEMM:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_PRECOMP_GEMM:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_GEMM:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_DIRECT:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_FFT;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT_TILING:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT_WINOGRAD:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;
			break;
		default: // -1: Using preferences to find a suitable algorithm
			hipdnnGetConvolutionForwardAlgorithm(cudnn, a_desc, w_desc, conv_desc, b_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);
	}

	size_t workspace_size = 0;
	hipdnnGetConvolutionForwardWorkspaceSize(cudnn, a_desc, w_desc, conv_desc, b_desc, algo, &workspace_size);
	void* workspace = 0;
	// TODO: If error, return OOM
	if (workspace_size)
		hipMalloc(&workspace, workspace_size);
	float one = 1, zero = 0;
	hipdnnConvolutionForward(cudnn, &one, a_desc, a->data.u8, w_desc, w->data.u8, conv_desc, algo, workspace, workspace_size, &zero, b_desc, b->data.u8);
	hipdnnAddTensor(cudnn, &one, bias_desc, bias->data.u8, &one, b_desc, b->data.u8);

	hipdnnDestroyTensorDescriptor(a_desc);
	hipdnnDestroyFilterDescriptor(w_desc);
	hipdnnDestroyTensorDescriptor(bias_desc);
	hipdnnDestroyTensorDescriptor(b_desc);
	hipdnnDestroyConvolutionDescriptor(conv_desc);
	if (workspace)
		hipFreeAsync(workspace, stream);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_conv_forw_autotune(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* inputs, const int input_size, ccv_nnc_tensor_t** outputs, const int output_size, const ccv_nnc_stream_context_t* stream_context)
{
	assert(input_size == 3);
	assert(output_size == 1);
	assert(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	hipdnnTensorDescriptor_t a_desc, bias_desc, b_desc;
	hipdnnFilterDescriptor_t w_desc;
	hipdnnCreateTensorDescriptor(&a_desc);
	const ccv_nnc_tensor_t* a = inputs[0];
	_ccv_nnc_set_tensor_nd_desc((const ccv_nnc_tensor_view_t*)a, a_desc);
	hipdnnCreateFilterDescriptor(&w_desc);
	const ccv_nnc_tensor_t* w = inputs[1];
	_ccv_nnc_set_filter_nd_desc(w, w_desc);
	hipdnnCreateTensorDescriptor(&bias_desc);
	const ccv_nnc_tensor_t* bias = inputs[2];
	_ccv_nnc_set_tensor_nd_desc((const ccv_nnc_tensor_view_t*)bias, bias_desc);
	hipdnnCreateTensorDescriptor(&b_desc);
	ccv_nnc_tensor_t* b = outputs[0];
	_ccv_nnc_set_tensor_nd_desc((const ccv_nnc_tensor_view_t*)b, b_desc);
	hipdnnConvolutionDescriptor_t conv_desc;
	hipdnnCreateConvolutionDescriptor(&conv_desc);
	int i;
	int p[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		p[i] = ccv_max(hint.border.begin[CCV_NNC_MAX_DIM - i], hint.border.end[CCV_NNC_MAX_DIM - i]);
	int v[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		v[i] = hint.stride.dim[CCV_NNC_MAX_DIM - i];
	int u[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		u[i] = 1;
	hipdnnSetConvolutionNdDescriptor(conv_desc, CCV_NNC_MAX_DIM, p, v, u, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

	int returnedAlgoCount;
	hipdnnConvolutionFwdAlgoPerf_t perfResults[CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT];
	hipdnnFindConvolutionForwardAlgorithm(cudnn, a_desc, w_desc, conv_desc, b_desc, CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT, &returnedAlgoCount, perfResults);
	for(int algoIndex = 0; algoIndex < returnedAlgoCount; ++algoIndex){
		printf("^^^^ %s for Algo %d: %f time requiring %llu memory\n", hipdnnGetErrorString(perfResults[algoIndex].status), perfResults[algoIndex].algo, perfResults[algoIndex].time, (unsigned long long)perfResults[algoIndex].memory);
	}

	hipdnnDestroyTensorDescriptor(a_desc);
	hipdnnDestroyFilterDescriptor(w_desc);
	hipdnnDestroyTensorDescriptor(bias_desc);
	hipdnnDestroyTensorDescriptor(b_desc);
	hipdnnDestroyConvolutionDescriptor(conv_desc);
	return -1; // Return the most efficient algorithm, return -1 if cannot find one.
}
#endif

//@ccv_nnc_init CCV_NNC_BACKEND_GPU_CUDNN
void ccv_nnc_gpu_cudnn_init(ccv_nnc_cmd_api_t cmd_api[])
{
#ifdef HAVE_CUDNN
	/* Convolutional layer */
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].tensor_memory = CCV_TENSOR_GPU_MEMORY;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].algorithms = CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].exec = _ccv_nnc_conv_forw;
	cmd_api[CCV_NNC_COMPUTE_CONVOLUTIONAL_FORWARD].autotune = _ccv_nnc_conv_forw_autotune;
	/* Full connect layer */
	/* Max pool layer */
	/* Average pool layer */
	/* Softmax layer */
	/* ReLU activation */
#endif
}
